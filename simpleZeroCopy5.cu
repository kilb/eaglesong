#include "hip/hip_runtime.h"
﻿/// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <stddef.h>
#include <time.h>

#define threadsPerBlock  (512)
#define MaxHashNum (1<<30)
#define MaxGpuNum (1024)
#define HashNum (1<<30)

#define MaxMesageLen (128)
#define Hash_Rounds (43)
#define Hash_Capacity (256)
#define Hash_Rate (256)
#define Hash_Len (32)
#define MaxEP_num (MaxMesageLen/(Hash_Rate>>5))
#define Data_Len (60)

#define ROL32(a,b) (((a)<<(b))|((a)>>(32-(b))))
#define ROL_ADD(a,b) a += b; a = ROL32(a, 8); b = ROL32(b, 24) + a;

#define ep_sta(u) \
{ \
	temp = stat0 ^ stat4 ^ stat12 ^ stat15; stat0 = temp^stat5 ^ stat6 ^ stat7; stat1 = temp^stat1 ^ stat8 ^ stat13; \
	temp = stat1 ^ stat2 ^ stat6 ^ stat14; stat2 = temp^stat7 ^ stat8 ^ stat9; stat3 = temp^stat3 ^ stat10 ^ stat15; \
	temp = stat0 ^ stat3 ^ stat4 ^ stat8; stat4 = temp^stat9 ^ stat10 ^ stat11; stat5 = temp^stat1 ^ stat5 ^ stat12; \
	temp = stat2 ^ stat5 ^ stat6 ^ stat10; stat6 = temp^stat11 ^ stat12 ^ stat13; stat7 = temp^stat3 ^ stat7 ^ stat14; \
	temp = stat4 ^ stat7 ^ stat8 ^ stat12; stat8 = temp^stat13 ^ stat14 ^ stat15; stat9 = temp^stat0 ^ stat5 ^ stat9; \
	temp = stat6 ^ stat9 ^ stat10 ^ stat14; stat10 = temp^stat0 ^ stat1 ^ stat15; stat11 = temp^stat2 ^ stat7 ^ stat11; \
	temp = stat0 ^ stat8 ^ stat11 ^ stat12; stat12 = temp^stat1 ^ stat2 ^ stat3; stat13 = temp^stat4 ^ stat9 ^ stat13; \
	temp = stat3 ^ stat5 ^ stat13 ^ stat14; stat14 = temp^stat2 ^ stat4 ^ stat10; stat15 = temp^stat0 ^ stat1 ^ stat6 ^ stat7 ^ stat8 ^ stat9 ^ stat15; \
	stat0 ^= ROL32(stat0, 2) ^ ROL32(stat0, 4) ^ gpu_injection_constants[(u + 0)];                    \
	stat1 ^= ROL32(stat1, 13) ^ ROL32(stat1, 22) ^ gpu_injection_constants[(u + 1)];                  \
	ROL_ADD(stat0, stat1);                                                                      \
	stat2 ^= ROL32(stat2, 4) ^ ROL32(stat2, 19) ^ gpu_injection_constants[(u + 2)];                   \
	stat3 ^= ROL32(stat3, 3) ^ ROL32(stat3, 14) ^ gpu_injection_constants[(u + 3)];                   \
	ROL_ADD(stat2, stat3);                                                                      \
	stat4 ^= ROL32(stat4, 27) ^ ROL32(stat4, 31) ^ gpu_injection_constants[(u + 4)];                  \
	stat5 ^= ROL32(stat5, 3) ^ ROL32(stat5, 8) ^ gpu_injection_constants[(u + 5)];                    \
	ROL_ADD(stat4, stat5);                                                                      \
	stat6 ^= ROL32(stat6, 17) ^ ROL32(stat6, 26) ^ gpu_injection_constants[(u + 6)];                  \
	stat7 ^= ROL32(stat7, 3) ^ ROL32(stat7, 12) ^ gpu_injection_constants[(u + 7)];                   \
	ROL_ADD(stat6, stat7);                                                                      \
	stat8 ^= ROL32(stat8, 18) ^ ROL32(stat8, 22) ^ gpu_injection_constants[(u + 8)];                  \
	stat9 ^= ROL32(stat9, 12) ^ ROL32(stat9, 18) ^ gpu_injection_constants[(u + 9)];                  \
	ROL_ADD(stat8, stat9);                                                                      \
	stat10 ^= ROL32(stat10, 4) ^ ROL32(stat10, 7) ^ gpu_injection_constants[(u + 10)];                 \
	stat11 ^= ROL32(stat11, 4) ^ ROL32(stat11, 31) ^ gpu_injection_constants[(u + 11)];                \
	ROL_ADD(stat10, stat11);                                                                    \
	stat12 ^= ROL32(stat12, 12) ^ ROL32(stat12, 27) ^ gpu_injection_constants[(u + 12)];               \
	stat13 ^= ROL32(stat13, 7) ^ ROL32(stat13, 17) ^ gpu_injection_constants[(u + 13)];                \
	ROL_ADD(stat12, stat13);                                                                    \
	stat14 ^= ROL32(stat14, 7) ^ ROL32(stat14, 8) ^ gpu_injection_constants[(u + 14)];                 \
	stat15 ^= ROL32(stat15, 1) ^ ROL32(stat15, 13) ^ gpu_injection_constants[(u + 15)];                \
	ROL_ADD(stat14, stat15); \
}

__constant__ uint32_t gpu_injection_constants[688] = //16*43 = 2752B，问题是这个超过了一个内存页，这意味着会很慢，运行过程会切换内存页
{
	0x6e9e40ae, 0x71927c02, 0x9a13d3b1, 0xdaec32ad, 0x3d8951cf, 0xe1c9fe9a, 0xb806b54c, 0xacbbf417,
	0xd3622b3b, 0xa082762a, 0x9edcf1c0, 0xa9bada77, 0x7f91e46c, 0xcb0f6e4f, 0x265d9241, 0xb7bdeab0,
	0x6260c9e6, 0xff50dd2a, 0x9036aa71, 0xce161879, 0xd1307cdf, 0x89e456df, 0xf83133e2, 0x65f55c3d,
	0x94871b01, 0xb5d204cd, 0x583a3264, 0x5e165957, 0x4cbda964, 0x675fca47, 0xf4a3033e, 0x2a417322,
	0x3b61432f, 0x7f5532f2, 0xb609973b, 0x1a795239, 0x31b477c9, 0xd2949d28, 0x78969712, 0x0eb87b6e,
	0x7e11d22d, 0xccee88bd, 0xeed07eb8, 0xe5563a81, 0xe7cb6bcf, 0x25de953e, 0x4d05653a, 0x0b831557,
	0x94b9cd77, 0x13f01579, 0x794b4a4a, 0x67e7c7dc, 0xc456d8d4, 0x59689c9b, 0x668456d7, 0x22d2a2e1,
	0x38b3a828, 0x0315ac3c, 0x438d681e, 0xab7109c5, 0x97ee19a8, 0xde062b2e, 0x2c76c47b, 0x0084456f,
	0x908f0fd3, 0xa646551f, 0x3e826725, 0xd521788e, 0x9f01c2b0, 0x93180cdc, 0x92ea1df8, 0x431a9aae,
	0x7c2ea356, 0xda33ad03, 0x46926893, 0x66bde7d7, 0xb501cc75, 0x1f6e8a41, 0x685250f4, 0x3bb1f318,
	0xaf238c04, 0x974ed2ec, 0x5b159e49, 0xd526f8bf, 0x12085626, 0x3e2432a9, 0x6bd20c48, 0x1f1d59da,
	0x18ab1068, 0x80f83cf8, 0x2c8c11c0, 0x7d548035, 0x0ff675c3, 0xfed160bf, 0x74bbbb24, 0xd98e006b,
	0xdeaa47eb, 0x05f2179e, 0x437b0b71, 0xa7c95f8f, 0x00a99d3b, 0x3fc3c444, 0x72686f8e, 0x00fd01a9,
	0xdedc0787, 0xc6af7626, 0x7012fe76, 0xf2a5f7ce, 0x9a7b2eda, 0x5e57fcf2, 0x4da0d4ad, 0x5c63b155,
	0x34117375, 0xd4134c11, 0x2ea77435, 0x5278b6de, 0xab522c4c, 0xbc8fc702, 0xc94a09e4, 0xebb93a9e,
	0x91ecb65e, 0x4c52ecc6, 0x8703bb52, 0xcb2d60aa, 0x30a0538a, 0x1514f10b, 0x157f6329, 0x3429dc3d,
	0x5db73eb2, 0xa7a1a969, 0x7286bd24, 0x0df6881e, 0x3785ba5f, 0xcd04623a, 0x02758170, 0xd827f556,
	0x99d95191, 0x84457eb1, 0x58a7fb22, 0xd2967c5f, 0x4f0c33f6, 0x4a02099a, 0xe0904821, 0x94124036,
	0x496a031b, 0x780b69c4, 0xcf1a4927, 0x87a119b8, 0xcdfaf4f8, 0x4cf9cd0f, 0x27c96a84, 0x6d11117e,
	0x7f8cf847, 0x74ceede5, 0xc88905e6, 0x60215841, 0x7172875a, 0x736e993a, 0x010aa53c, 0x43d53c2b,
	0xf0d91a93, 0x0d983b56, 0xf816663c, 0xe5d13363, 0x0a61737c, 0x09d51150, 0x83a5ac2f, 0x3e884905,
	0x7b01aeb5, 0x600a6ea7, 0xb7678f7b, 0x72b38977, 0x068018f2, 0xce6ae45b, 0x29188aa8, 0xe5a0b1e9,
	0xc04c2b86, 0x8bd14d75, 0x648781f3, 0xdbae1e0a, 0xddcdd8ae, 0xab4d81a3, 0x446baaba, 0x1cc0c19d,
	0x17be4f90, 0x82c0e65d, 0x676f9c95, 0x5c708db2, 0x6fd4c867, 0xa5106ef0, 0x19dde49d, 0x78182f95,
	0xd089cd81, 0xa32e98fe, 0xbe306c82, 0x6cd83d8c, 0x037f1bde, 0x0b15722d, 0xeddc1e22, 0x93c76559,
	0x8a2f571b, 0x92cc81b4, 0x021b7477, 0x67523904, 0xc95dbccc, 0xac17ee9d, 0x944e46bc, 0x0781867e,
	0xc854dd9d, 0x26e2c30c, 0x858c0416, 0x6d397708, 0xebe29c58, 0xc80ced86, 0xd496b4ab, 0xbe45e6f5,
	0x10d24706, 0xacf8187a, 0x96f523cb, 0x2227e143, 0x78c36564, 0x4643adc2, 0x4729d97a, 0xcff93e0d,
	0x25484bbd, 0x91c6798e, 0x95f773f4, 0x44204675, 0x2eda57ba, 0x06d313ef, 0xeeaa4466, 0x2dfa7530,
	0xa8af0c9b, 0x39f1535e, 0x0cc2b7bd, 0x38a76c0e, 0x4f41071d, 0xcdaf2475, 0x49a6eff8, 0x01621748,
	0x36ebacab, 0xbd6d9a29, 0x44d1cd65, 0x40815dfd, 0x55fa5a1a, 0x87cce9e9, 0xae559b45, 0xd76b4c26,
	0x637d60ad, 0xde29f5f9, 0x97491cbb, 0xfb350040, 0xffe7f997, 0x201c9dcd, 0xe61320e9, 0xa90987a3,
	0xe24afa83, 0x61c1e6fc, 0xcc87ff62, 0xf1c9d8fa, 0x4fd04546, 0x90ecc76e, 0x46e456b9, 0x305dceb8,
	0xf627e68c, 0x2d286815, 0xc705bbfd, 0x101b6df3, 0x892dae62, 0xd5b7fb44, 0xea1d5c94, 0x5332e3cb,
	0xf856f88a, 0xb341b0e9, 0x28408d9d, 0x5421bc17, 0xeb9af9bc, 0x602371c5, 0x67985a91, 0xd774907f,
	0x7c4d697d, 0x9370b0b8, 0x6ff5cebb, 0x7d465744, 0x674ceac0, 0xea9102fc, 0x0de94784, 0xc793de69,
	0xfe599bb1, 0xc6ad952f, 0x6d6ca9c3, 0x928c3f91, 0xf9022f05, 0x24a164dc, 0xe5e98cd3, 0x7649efdb,
	0x6df3bcdb, 0x5d1e9ff1, 0x17f5d010, 0xe2686ea1, 0x6eac77fe, 0x7bb5c585, 0x88d90cbb, 0x18689163,
	0x67c9efa5, 0xc0b76d9b, 0x960efbab, 0xbd872807, 0x70f4c474, 0x56c29d20, 0xd1541d15, 0x88137033,
	0xe3f02b3e, 0xb6d9b28d, 0x53a077ba, 0xeedcd29e, 0xa50a6c1d, 0x12c2801e, 0x52ba335b, 0x35984614,
	0xe2599aa8, 0xaf94ed1d, 0xd90d4767, 0x202c7d07, 0x77bec4f4, 0xfa71bc80, 0xfc5c8b76, 0x8d0fbbfc,
	0xda366dc6, 0x8b32a0c7, 0x1b36f7fc, 0x6642dcbc, 0x6fe7e724, 0x8b5fa782, 0xc4227404, 0x3a7d1da7,
	0x517ed658, 0x8a18df6d, 0x3e5c9b23, 0x1fbd51ef, 0x1470601d, 0x3400389c, 0x676b065d, 0x8864ad80,
	0xea6f1a9c, 0x2db484e1, 0x608785f0, 0x8dd384af, 0x69d26699, 0x409c4e16, 0x77f9986a, 0x7f491266,
	0x883ea6cf, 0xeaa06072, 0xfa2e5db5, 0x352594b4, 0x9156bb89, 0xa2fbbbfb, 0xac3989c7, 0x6e2422b1,
	0x581f3560, 0x1009a9b5, 0x7e5ad9cd, 0xa9fc0a6e, 0x43e5998e, 0x7f8778f9, 0xf038f8e1, 0x5415c2e8,
	0x6499b731, 0xb82389ae, 0x05d4d819, 0x0f06440e, 0xf1735aa0, 0x986430ee, 0x47ec952c, 0xbf149cc5,
	0xb3cb2cb6, 0x3f41e8c2, 0x271ac51b, 0x48ac5ded, 0xf76a0469, 0x717bba4d, 0x4f5c90d6, 0x3b74f756,
	0x1824110a, 0xa4fd43e3, 0x1eb0507c, 0xa9375c08, 0x157c59a7, 0x0cad8f51, 0xd66031a0, 0xabb5343f,
	0xe533fa43, 0x1996e2bb, 0xd7953a71, 0xd2529b94, 0x58f0fa07, 0x4c9b1877, 0x057e990d, 0x8bfe19c4,
	0xa8e2c0c9, 0x99fcaada, 0x69d2aaca, 0xdc1c4642, 0xf4d22307, 0x7fe27e8c, 0x1366aa07, 0x1594e637,
	0xce1066bf, 0xdb922552, 0x9930b52a, 0xaeaa9a3e, 0x31ff7eb4, 0x5e1f945a, 0x150ac49c, 0x0ccdac2d,
	0xd8a8a217, 0xb82ea6e5, 0xd6a74659, 0x67b7e3e6, 0x836eef4a, 0xb6f90074, 0x7fa3ea4b, 0xcb038123,
	0xbf069f55, 0x1fa83fc4, 0xd6ebdb23, 0x16f0a137, 0x19a7110d, 0x5ff3b55f, 0xfb633868, 0xb466f845,
	0xbce0c198, 0x88404296, 0xddbdd88b, 0x7fc52546, 0x63a553f8, 0xa728405a, 0x378a2bce, 0x6862e570,
	0xefb77e7d, 0xc611625e, 0x32515c15, 0x6984b765, 0xe8405976, 0x9ba386fd, 0xd4eed4d9, 0xf8fe0309,
	0x0ce54601, 0xbaf879c2, 0xd8524057, 0x1d8c1d7a, 0x72c0a3a9, 0x5a1ffbde, 0x82f33a45, 0x5143f446,
	0x29c7e182, 0xe536c32f, 0x5a6f245b, 0x44272adb, 0xcb701d9c, 0xf76137ec, 0x0841f145, 0xe7042ecc,
	0xf1277dd7, 0x745cf92c, 0xa8fe65fe, 0xd3e2d7cf, 0x54c513ef, 0x6079bc2d, 0xb66336b0, 0x101e383b,
	0xbcd75753, 0x25be238a, 0x56a6f0be, 0xeeffcc17, 0x5ea31f3d, 0x0ae772f5, 0xf76de3de, 0x1bbecdad,
	0xc9107d43, 0xf7e38dce, 0x618358cd, 0x5c833f04, 0xf6975906, 0xde4177e5, 0x67d314dc, 0xb4760f3e,
	0x56ce5888, 0x0e8345a8, 0xbff6b1bf, 0x78dfb112, 0xf1709c1e, 0x7bb8ed8b, 0x902402b9, 0xdaa64ae0,
	0x46b71d89, 0x7eee035f, 0xbe376509, 0x99648f3a, 0x0863ea1f, 0x49ad8887, 0x79bdecc5, 0x3c10b568,
	0x5f2e4bae, 0x04ef20ab, 0x72f8ce7b, 0x521e1ebe, 0x14525535, 0x2e8af95b, 0x9094ccfd, 0xbcf36713,
	0xc73953ef, 0xd4b91474, 0x6554ec2d, 0xe3885c96, 0x03dc73b7, 0x931688a9, 0xcbbef182, 0x2b77cfc9,
	0x632a32bd, 0xd2115dcc, 0x1ae5533d, 0x32684e13, 0x4cc5a004, 0x13321bde, 0x62cbd38d, 0x78383a3b,
	0xd00686f1, 0x9f601ee7, 0x7eaf23de, 0x3110c492, 0x9c351209, 0x7eb89d52, 0x6d566eac, 0xc2efd226,
	0x32e9fac5, 0x52227274, 0x09f84725, 0xb8d0b605, 0x72291f02, 0x71b5c34b, 0x3dbfcbb8, 0x04a02263,
	0x55ba597f, 0xd4e4037d, 0xc813e1be, 0xffddeefa, 0xc3c058f3, 0x87010f2e, 0x1dfcf55f, 0xc694eeeb,
	0xa9c01a74, 0x98c2fc6b, 0xe57e1428, 0xdd265a71, 0x836b956d, 0x7e46ab1a, 0x5835d541, 0x50b32505,
	0xe640913c, 0xbb486079, 0xfe496263, 0x113c5b69, 0x93cd6620, 0x5efe823b, 0x2d657b40, 0xb46dfc6c,
	0x57710c69, 0xfe9fadeb, 0xb5f8728a, 0xe3224170, 0xca28b751, 0xfdabae56, 0x5ab12c3c, 0xa697c457,
	0xd28fa2b7, 0x056579f2, 0x9fd9d810, 0xe3557478, 0xd88d89ab, 0xa72a9422, 0x6d47abd0, 0x405bcbd9,
	0x6f83ebaf, 0x13caec76, 0xfceb9ee2, 0x2e922df7, 0xce9856df, 0xc05e9322, 0x2772c854, 0xb67f2a32,
	0x6d1af28d, 0x3a78cf77, 0xdff411e4, 0x61c74ca9, 0xed8b842e, 0x72880845, 0x6e857085, 0xc6404932,
	0xee37f6bc, 0x27116f48, 0x5e9ec45a, 0x8ea2a51f, 0xa5573db7, 0xa746d036, 0x486b4768, 0x5b438f3b,
	0x18c54a5c, 0x64fcf08e, 0xe993cdc1, 0x35c1ead3, 0x9de07de7, 0x321b841c, 0x87423c5e, 0x071aa0f6,
	0x962eb75b, 0xbb06bdd2, 0xdcdb5363, 0x389752f2, 0x83d9cc88, 0xd014adc6, 0xc71121bb, 0x2372f938,
	0xcaff2650, 0x62be8951, 0x56dccaff, 0xac4084c0, 0x09712e95, 0x1d3c288f, 0x1b085744, 0xe1d3cfef,
	0x5c9a812e, 0x6611fd59, 0x85e46044, 0x1981d885, 0x5a4c903f, 0x43f30d4b, 0x7d1d601b, 0xdd3c3391,
	0x030ec65e, 0xc12878cd, 0x72e795fe, 0xd0c76abd, 0x1ec085db, 0x7cbb61fa, 0x93e8dd1e, 0x8582eb06,
	0x73563144, 0x049d4e7e, 0x5fd5aefe, 0x7b842a00, 0x75ced665, 0xbb32d458, 0x4e83bba7, 0x8f15151f,
	0x7795a125, 0xf0842455, 0x499af99d, 0x565cc7fa, 0xa3b1278d, 0x3f27ce74, 0x96ca058e, 0x8a497443,
	0xa6fb8cae, 0xc115aa21, 0x17504923, 0xe4932402, 0xaea886c2, 0x8eb79af5, 0xebd5ea6b, 0xc7980d3b,
	0x71369315, 0x796e6a66, 0x3a7ec708, 0xb05175c8, 0xe02b74e7, 0xeb377ad3, 0x6c8c1f54, 0xb980c374,
	0x59aee281, 0x449cb799, 0xe01f5605, 0xed0e085e, 0xc9a1a3b4, 0xaac481b1, 0xc935c39c, 0xb7d8ce7f
};
const uint32_t injection_constants[688] = //16*43 = 2752B，问题是这个超过了一个内存页，这意味着会很慢，运行过程会切换内存页
{
	0x6e9e40ae, 0x71927c02, 0x9a13d3b1, 0xdaec32ad, 0x3d8951cf, 0xe1c9fe9a, 0xb806b54c, 0xacbbf417,
	0xd3622b3b, 0xa082762a, 0x9edcf1c0, 0xa9bada77, 0x7f91e46c, 0xcb0f6e4f, 0x265d9241, 0xb7bdeab0,
	0x6260c9e6, 0xff50dd2a, 0x9036aa71, 0xce161879, 0xd1307cdf, 0x89e456df, 0xf83133e2, 0x65f55c3d,
	0x94871b01, 0xb5d204cd, 0x583a3264, 0x5e165957, 0x4cbda964, 0x675fca47, 0xf4a3033e, 0x2a417322,
	0x3b61432f, 0x7f5532f2, 0xb609973b, 0x1a795239, 0x31b477c9, 0xd2949d28, 0x78969712, 0x0eb87b6e,
	0x7e11d22d, 0xccee88bd, 0xeed07eb8, 0xe5563a81, 0xe7cb6bcf, 0x25de953e, 0x4d05653a, 0x0b831557,
	0x94b9cd77, 0x13f01579, 0x794b4a4a, 0x67e7c7dc, 0xc456d8d4, 0x59689c9b, 0x668456d7, 0x22d2a2e1,
	0x38b3a828, 0x0315ac3c, 0x438d681e, 0xab7109c5, 0x97ee19a8, 0xde062b2e, 0x2c76c47b, 0x0084456f,
	0x908f0fd3, 0xa646551f, 0x3e826725, 0xd521788e, 0x9f01c2b0, 0x93180cdc, 0x92ea1df8, 0x431a9aae,
	0x7c2ea356, 0xda33ad03, 0x46926893, 0x66bde7d7, 0xb501cc75, 0x1f6e8a41, 0x685250f4, 0x3bb1f318,
	0xaf238c04, 0x974ed2ec, 0x5b159e49, 0xd526f8bf, 0x12085626, 0x3e2432a9, 0x6bd20c48, 0x1f1d59da,
	0x18ab1068, 0x80f83cf8, 0x2c8c11c0, 0x7d548035, 0x0ff675c3, 0xfed160bf, 0x74bbbb24, 0xd98e006b,
	0xdeaa47eb, 0x05f2179e, 0x437b0b71, 0xa7c95f8f, 0x00a99d3b, 0x3fc3c444, 0x72686f8e, 0x00fd01a9,
	0xdedc0787, 0xc6af7626, 0x7012fe76, 0xf2a5f7ce, 0x9a7b2eda, 0x5e57fcf2, 0x4da0d4ad, 0x5c63b155,
	0x34117375, 0xd4134c11, 0x2ea77435, 0x5278b6de, 0xab522c4c, 0xbc8fc702, 0xc94a09e4, 0xebb93a9e,
	0x91ecb65e, 0x4c52ecc6, 0x8703bb52, 0xcb2d60aa, 0x30a0538a, 0x1514f10b, 0x157f6329, 0x3429dc3d,
	0x5db73eb2, 0xa7a1a969, 0x7286bd24, 0x0df6881e, 0x3785ba5f, 0xcd04623a, 0x02758170, 0xd827f556,
	0x99d95191, 0x84457eb1, 0x58a7fb22, 0xd2967c5f, 0x4f0c33f6, 0x4a02099a, 0xe0904821, 0x94124036,
	0x496a031b, 0x780b69c4, 0xcf1a4927, 0x87a119b8, 0xcdfaf4f8, 0x4cf9cd0f, 0x27c96a84, 0x6d11117e,
	0x7f8cf847, 0x74ceede5, 0xc88905e6, 0x60215841, 0x7172875a, 0x736e993a, 0x010aa53c, 0x43d53c2b,
	0xf0d91a93, 0x0d983b56, 0xf816663c, 0xe5d13363, 0x0a61737c, 0x09d51150, 0x83a5ac2f, 0x3e884905,
	0x7b01aeb5, 0x600a6ea7, 0xb7678f7b, 0x72b38977, 0x068018f2, 0xce6ae45b, 0x29188aa8, 0xe5a0b1e9,
	0xc04c2b86, 0x8bd14d75, 0x648781f3, 0xdbae1e0a, 0xddcdd8ae, 0xab4d81a3, 0x446baaba, 0x1cc0c19d,
	0x17be4f90, 0x82c0e65d, 0x676f9c95, 0x5c708db2, 0x6fd4c867, 0xa5106ef0, 0x19dde49d, 0x78182f95,
	0xd089cd81, 0xa32e98fe, 0xbe306c82, 0x6cd83d8c, 0x037f1bde, 0x0b15722d, 0xeddc1e22, 0x93c76559,
	0x8a2f571b, 0x92cc81b4, 0x021b7477, 0x67523904, 0xc95dbccc, 0xac17ee9d, 0x944e46bc, 0x0781867e,
	0xc854dd9d, 0x26e2c30c, 0x858c0416, 0x6d397708, 0xebe29c58, 0xc80ced86, 0xd496b4ab, 0xbe45e6f5,
	0x10d24706, 0xacf8187a, 0x96f523cb, 0x2227e143, 0x78c36564, 0x4643adc2, 0x4729d97a, 0xcff93e0d,
	0x25484bbd, 0x91c6798e, 0x95f773f4, 0x44204675, 0x2eda57ba, 0x06d313ef, 0xeeaa4466, 0x2dfa7530,
	0xa8af0c9b, 0x39f1535e, 0x0cc2b7bd, 0x38a76c0e, 0x4f41071d, 0xcdaf2475, 0x49a6eff8, 0x01621748,
	0x36ebacab, 0xbd6d9a29, 0x44d1cd65, 0x40815dfd, 0x55fa5a1a, 0x87cce9e9, 0xae559b45, 0xd76b4c26,
	0x637d60ad, 0xde29f5f9, 0x97491cbb, 0xfb350040, 0xffe7f997, 0x201c9dcd, 0xe61320e9, 0xa90987a3,
	0xe24afa83, 0x61c1e6fc, 0xcc87ff62, 0xf1c9d8fa, 0x4fd04546, 0x90ecc76e, 0x46e456b9, 0x305dceb8,
	0xf627e68c, 0x2d286815, 0xc705bbfd, 0x101b6df3, 0x892dae62, 0xd5b7fb44, 0xea1d5c94, 0x5332e3cb,
	0xf856f88a, 0xb341b0e9, 0x28408d9d, 0x5421bc17, 0xeb9af9bc, 0x602371c5, 0x67985a91, 0xd774907f,
	0x7c4d697d, 0x9370b0b8, 0x6ff5cebb, 0x7d465744, 0x674ceac0, 0xea9102fc, 0x0de94784, 0xc793de69,
	0xfe599bb1, 0xc6ad952f, 0x6d6ca9c3, 0x928c3f91, 0xf9022f05, 0x24a164dc, 0xe5e98cd3, 0x7649efdb,
	0x6df3bcdb, 0x5d1e9ff1, 0x17f5d010, 0xe2686ea1, 0x6eac77fe, 0x7bb5c585, 0x88d90cbb, 0x18689163,
	0x67c9efa5, 0xc0b76d9b, 0x960efbab, 0xbd872807, 0x70f4c474, 0x56c29d20, 0xd1541d15, 0x88137033,
	0xe3f02b3e, 0xb6d9b28d, 0x53a077ba, 0xeedcd29e, 0xa50a6c1d, 0x12c2801e, 0x52ba335b, 0x35984614,
	0xe2599aa8, 0xaf94ed1d, 0xd90d4767, 0x202c7d07, 0x77bec4f4, 0xfa71bc80, 0xfc5c8b76, 0x8d0fbbfc,
	0xda366dc6, 0x8b32a0c7, 0x1b36f7fc, 0x6642dcbc, 0x6fe7e724, 0x8b5fa782, 0xc4227404, 0x3a7d1da7,
	0x517ed658, 0x8a18df6d, 0x3e5c9b23, 0x1fbd51ef, 0x1470601d, 0x3400389c, 0x676b065d, 0x8864ad80,
	0xea6f1a9c, 0x2db484e1, 0x608785f0, 0x8dd384af, 0x69d26699, 0x409c4e16, 0x77f9986a, 0x7f491266,
	0x883ea6cf, 0xeaa06072, 0xfa2e5db5, 0x352594b4, 0x9156bb89, 0xa2fbbbfb, 0xac3989c7, 0x6e2422b1,
	0x581f3560, 0x1009a9b5, 0x7e5ad9cd, 0xa9fc0a6e, 0x43e5998e, 0x7f8778f9, 0xf038f8e1, 0x5415c2e8,
	0x6499b731, 0xb82389ae, 0x05d4d819, 0x0f06440e, 0xf1735aa0, 0x986430ee, 0x47ec952c, 0xbf149cc5,
	0xb3cb2cb6, 0x3f41e8c2, 0x271ac51b, 0x48ac5ded, 0xf76a0469, 0x717bba4d, 0x4f5c90d6, 0x3b74f756,
	0x1824110a, 0xa4fd43e3, 0x1eb0507c, 0xa9375c08, 0x157c59a7, 0x0cad8f51, 0xd66031a0, 0xabb5343f,
	0xe533fa43, 0x1996e2bb, 0xd7953a71, 0xd2529b94, 0x58f0fa07, 0x4c9b1877, 0x057e990d, 0x8bfe19c4,
	0xa8e2c0c9, 0x99fcaada, 0x69d2aaca, 0xdc1c4642, 0xf4d22307, 0x7fe27e8c, 0x1366aa07, 0x1594e637,
	0xce1066bf, 0xdb922552, 0x9930b52a, 0xaeaa9a3e, 0x31ff7eb4, 0x5e1f945a, 0x150ac49c, 0x0ccdac2d,
	0xd8a8a217, 0xb82ea6e5, 0xd6a74659, 0x67b7e3e6, 0x836eef4a, 0xb6f90074, 0x7fa3ea4b, 0xcb038123,
	0xbf069f55, 0x1fa83fc4, 0xd6ebdb23, 0x16f0a137, 0x19a7110d, 0x5ff3b55f, 0xfb633868, 0xb466f845,
	0xbce0c198, 0x88404296, 0xddbdd88b, 0x7fc52546, 0x63a553f8, 0xa728405a, 0x378a2bce, 0x6862e570,
	0xefb77e7d, 0xc611625e, 0x32515c15, 0x6984b765, 0xe8405976, 0x9ba386fd, 0xd4eed4d9, 0xf8fe0309,
	0x0ce54601, 0xbaf879c2, 0xd8524057, 0x1d8c1d7a, 0x72c0a3a9, 0x5a1ffbde, 0x82f33a45, 0x5143f446,
	0x29c7e182, 0xe536c32f, 0x5a6f245b, 0x44272adb, 0xcb701d9c, 0xf76137ec, 0x0841f145, 0xe7042ecc,
	0xf1277dd7, 0x745cf92c, 0xa8fe65fe, 0xd3e2d7cf, 0x54c513ef, 0x6079bc2d, 0xb66336b0, 0x101e383b,
	0xbcd75753, 0x25be238a, 0x56a6f0be, 0xeeffcc17, 0x5ea31f3d, 0x0ae772f5, 0xf76de3de, 0x1bbecdad,
	0xc9107d43, 0xf7e38dce, 0x618358cd, 0x5c833f04, 0xf6975906, 0xde4177e5, 0x67d314dc, 0xb4760f3e,
	0x56ce5888, 0x0e8345a8, 0xbff6b1bf, 0x78dfb112, 0xf1709c1e, 0x7bb8ed8b, 0x902402b9, 0xdaa64ae0,
	0x46b71d89, 0x7eee035f, 0xbe376509, 0x99648f3a, 0x0863ea1f, 0x49ad8887, 0x79bdecc5, 0x3c10b568,
	0x5f2e4bae, 0x04ef20ab, 0x72f8ce7b, 0x521e1ebe, 0x14525535, 0x2e8af95b, 0x9094ccfd, 0xbcf36713,
	0xc73953ef, 0xd4b91474, 0x6554ec2d, 0xe3885c96, 0x03dc73b7, 0x931688a9, 0xcbbef182, 0x2b77cfc9,
	0x632a32bd, 0xd2115dcc, 0x1ae5533d, 0x32684e13, 0x4cc5a004, 0x13321bde, 0x62cbd38d, 0x78383a3b,
	0xd00686f1, 0x9f601ee7, 0x7eaf23de, 0x3110c492, 0x9c351209, 0x7eb89d52, 0x6d566eac, 0xc2efd226,
	0x32e9fac5, 0x52227274, 0x09f84725, 0xb8d0b605, 0x72291f02, 0x71b5c34b, 0x3dbfcbb8, 0x04a02263,
	0x55ba597f, 0xd4e4037d, 0xc813e1be, 0xffddeefa, 0xc3c058f3, 0x87010f2e, 0x1dfcf55f, 0xc694eeeb,
	0xa9c01a74, 0x98c2fc6b, 0xe57e1428, 0xdd265a71, 0x836b956d, 0x7e46ab1a, 0x5835d541, 0x50b32505,
	0xe640913c, 0xbb486079, 0xfe496263, 0x113c5b69, 0x93cd6620, 0x5efe823b, 0x2d657b40, 0xb46dfc6c,
	0x57710c69, 0xfe9fadeb, 0xb5f8728a, 0xe3224170, 0xca28b751, 0xfdabae56, 0x5ab12c3c, 0xa697c457,
	0xd28fa2b7, 0x056579f2, 0x9fd9d810, 0xe3557478, 0xd88d89ab, 0xa72a9422, 0x6d47abd0, 0x405bcbd9,
	0x6f83ebaf, 0x13caec76, 0xfceb9ee2, 0x2e922df7, 0xce9856df, 0xc05e9322, 0x2772c854, 0xb67f2a32,
	0x6d1af28d, 0x3a78cf77, 0xdff411e4, 0x61c74ca9, 0xed8b842e, 0x72880845, 0x6e857085, 0xc6404932,
	0xee37f6bc, 0x27116f48, 0x5e9ec45a, 0x8ea2a51f, 0xa5573db7, 0xa746d036, 0x486b4768, 0x5b438f3b,
	0x18c54a5c, 0x64fcf08e, 0xe993cdc1, 0x35c1ead3, 0x9de07de7, 0x321b841c, 0x87423c5e, 0x071aa0f6,
	0x962eb75b, 0xbb06bdd2, 0xdcdb5363, 0x389752f2, 0x83d9cc88, 0xd014adc6, 0xc71121bb, 0x2372f938,
	0xcaff2650, 0x62be8951, 0x56dccaff, 0xac4084c0, 0x09712e95, 0x1d3c288f, 0x1b085744, 0xe1d3cfef,
	0x5c9a812e, 0x6611fd59, 0x85e46044, 0x1981d885, 0x5a4c903f, 0x43f30d4b, 0x7d1d601b, 0xdd3c3391,
	0x030ec65e, 0xc12878cd, 0x72e795fe, 0xd0c76abd, 0x1ec085db, 0x7cbb61fa, 0x93e8dd1e, 0x8582eb06,
	0x73563144, 0x049d4e7e, 0x5fd5aefe, 0x7b842a00, 0x75ced665, 0xbb32d458, 0x4e83bba7, 0x8f15151f,
	0x7795a125, 0xf0842455, 0x499af99d, 0x565cc7fa, 0xa3b1278d, 0x3f27ce74, 0x96ca058e, 0x8a497443,
	0xa6fb8cae, 0xc115aa21, 0x17504923, 0xe4932402, 0xaea886c2, 0x8eb79af5, 0xebd5ea6b, 0xc7980d3b,
	0x71369315, 0x796e6a66, 0x3a7ec708, 0xb05175c8, 0xe02b74e7, 0xeb377ad3, 0x6c8c1f54, 0xb980c374,
	0x59aee281, 0x449cb799, 0xe01f5605, 0xed0e085e, 0xc9a1a3b4, 0xaac481b1, 0xc935c39c, 0xb7d8ce7f
};
void PrintState(uint32_t * state)
{
	int i;
	for (i = 0; i < 16; ++i)
	{
		printf("0x%02x%02x%02x%02x ", (state[i] >> (3 * 8)) & 0xff, (state[i] >> (2 * 8)) & 0xff, (state[i] >> (1 * 8)) & 0xff, (state[i] >> (0 * 8)) & 0xff);
	}
	printf("\n");
}

void EaglesongPermutation(uint32_t * state)
{
	uint32_t state2[16], temp0, temp1,i,j;
	//PrintState(state);
	for (i = 0; i < Hash_Rounds; ++i)
	{
		temp0 = 0;
		temp0 = temp0 ^ state[0];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[5];
		temp0 = temp0 ^ state[6];
		temp0 = temp0 ^ state[7];
		temp0 = temp0 ^ state[12];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 2) ^ ROL32(temp0, 4) ^ injection_constants[i * 16 + 0];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[1];
		temp1 = temp1 ^ state[4];
		temp1 = temp1 ^ state[8];
		temp1 = temp1 ^ state[12];
		temp1 = temp1 ^ state[13];
		temp1 = temp1 ^ state[15];
		temp1 = temp1 ^ ROL32(temp1, 13) ^ ROL32(temp1, 22) ^ injection_constants[i * 16 + 1];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[1 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[1] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state[0];
		temp0 = temp0 ^ state[1];
		temp0 = temp0 ^ state[2];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[6];
		temp0 = temp0 ^ state[7];
		temp0 = temp0 ^ state[9];
		temp0 = temp0 ^ state[12];
		temp0 = temp0 ^ state[13];
		temp0 = temp0 ^ state[14];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 4) ^ ROL32(temp0, 19) ^ injection_constants[i * 16 + 2];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[1];
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[3];
		temp1 = temp1 ^ state[4];
		temp1 = temp1 ^ state[6];
		temp1 = temp1 ^ state[8];
		temp1 = temp1 ^ state[10];
		temp1 = temp1 ^ state[12];
		temp1 = temp1 ^ state[13];
		temp1 = temp1 ^ state[14];
		temp1 = temp1 ^ ROL32(temp1, 3) ^ ROL32(temp1, 14) ^ injection_constants[i * 16 + 3];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[3 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[3] = temp0 + temp1;


		temp0 = 0;
		temp0 = temp0 ^ state[1];
		temp0 = temp0 ^ state[2];
		temp0 = temp0 ^ state[3];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[5];
		temp0 = temp0 ^ state[7];
		temp0 = temp0 ^ state[9];
		temp0 = temp0 ^ state[11];
		temp0 = temp0 ^ state[13];
		temp0 = temp0 ^ state[14];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 27) ^ ROL32(temp0, 31) ^ injection_constants[i * 16 + 4];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[3];
		temp1 = temp1 ^ state[7];
		temp1 = temp1 ^ state[8];
		temp1 = temp1 ^ state[10];
		temp1 = temp1 ^ state[14];
		temp1 = temp1 ^ ROL32(temp1, 3) ^ ROL32(temp1, 8) ^ injection_constants[i * 16 + 5];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[5 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[5] = temp0 + temp1;


		temp0 = 0;
		temp0 = temp0 ^ state[1];
		temp0 = temp0 ^ state[3];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[8];
		temp0 = temp0 ^ state[9];
		temp0 = temp0 ^ state[11];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 17) ^ ROL32(temp0, 26) ^ injection_constants[i * 16 + 6];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[6];
		temp1 = temp1 ^ state[7];
		temp1 = temp1 ^ state[9];
		temp1 = temp1 ^ state[10];
		temp1 = temp1 ^ state[15];
		temp1 = temp1 ^ ROL32(temp1, 3) ^ ROL32(temp1, 12) ^ injection_constants[i * 16 + 7];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[7 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[7] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state[0];
		temp0 = temp0 ^ state[1];
		temp0 = temp0 ^ state[3];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[5];
		temp0 = temp0 ^ state[6];
		temp0 = temp0 ^ state[8];
		temp0 = temp0 ^ state[10];
		temp0 = temp0 ^ state[11];
		temp0 = temp0 ^ state[12];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 18) ^ ROL32(temp0, 22) ^ injection_constants[i * 16 + 8];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[1];
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[9];
		temp1 = temp1 ^ state[11];
		temp1 = temp1 ^ state[13];
		temp1 = temp1 ^ state[15];
		temp1 = temp1 ^ ROL32(temp1, 12) ^ ROL32(temp1, 18) ^ injection_constants[i * 16 + 9];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[9 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[9] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state[0];
		temp0 = temp0 ^ state[1];
		temp0 = temp0 ^ state[2];
		temp0 = temp0 ^ state[3];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[5];
		temp0 = temp0 ^ state[6];
		temp0 = temp0 ^ state[7];
		temp0 = temp0 ^ state[10];
		temp0 = temp0 ^ state[14];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 4) ^ ROL32(temp0, 7) ^ injection_constants[i * 16 + 10];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[1];
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[3];
		temp1 = temp1 ^ state[8];
		temp1 = temp1 ^ state[11];
		temp1 = temp1 ^ state[12];
		temp1 = temp1 ^ ROL32(temp1, 4) ^ ROL32(temp1, 31) ^ injection_constants[i * 16 + 11];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[11 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[11] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state[1];
		temp0 = temp0 ^ state[2];
		temp0 = temp0 ^ state[3];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[9];
		temp0 = temp0 ^ state[12];
		temp0 = temp0 ^ state[13];
		temp0 = temp0 ^ ROL32(temp0, 12) ^ ROL32(temp0, 27) ^ injection_constants[i * 16 + 12];

		temp1 = 0;
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[3];
		temp1 = temp1 ^ state[4];
		temp1 = temp1 ^ state[5];
		temp1 = temp1 ^ state[10];
		temp1 = temp1 ^ state[13];
		temp1 = temp1 ^ state[14];
		temp1 = temp1 ^ ROL32(temp1, 7) ^ ROL32(temp1, 17) ^ injection_constants[i * 16 + 13];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[13 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[13] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state[3];
		temp0 = temp0 ^ state[4];
		temp0 = temp0 ^ state[5];
		temp0 = temp0 ^ state[6];
		temp0 = temp0 ^ state[11];
		temp0 = temp0 ^ state[14];
		temp0 = temp0 ^ state[15];
		temp0 = temp0 ^ ROL32(temp0, 7) ^ ROL32(temp0, 8) ^ injection_constants[i * 16 + 14];

		temp1 = 0;
		temp1 = temp1 ^ state[0];
		temp1 = temp1 ^ state[1];
		temp1 = temp1 ^ state[2];
		temp1 = temp1 ^ state[3];
		temp1 = temp1 ^ state[5];
		temp1 = temp1 ^ state[7];
		temp1 = temp1 ^ state[8];
		temp1 = temp1 ^ state[9];
		temp1 = temp1 ^ state[10];
		temp1 = temp1 ^ state[11];
		temp1 = temp1 ^ state[15];
		temp1 = temp1 ^ ROL32(temp1, 1) ^ ROL32(temp1, 13) ^ injection_constants[i * 16 + 15];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state2[15 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state2[15] = temp0 + temp1;

		i++;
		if (i >= Hash_Rounds)break;

		temp0 = 0;
		temp0 = temp0 ^ state2[0];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[5];
		temp0 = temp0 ^ state2[6];
		temp0 = temp0 ^ state2[7];
		temp0 = temp0 ^ state2[12];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 2) ^ ROL32(temp0, 4) ^ injection_constants[i * 16 + 0];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[1];
		temp1 = temp1 ^ state2[4];
		temp1 = temp1 ^ state2[8];
		temp1 = temp1 ^ state2[12];
		temp1 = temp1 ^ state2[13];
		temp1 = temp1 ^ state2[15];
		temp1 = temp1 ^ ROL32(temp1, 13) ^ ROL32(temp1, 22) ^ injection_constants[i * 16 + 1];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[1 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[1] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state2[0];
		temp0 = temp0 ^ state2[1];
		temp0 = temp0 ^ state2[2];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[6];
		temp0 = temp0 ^ state2[7];
		temp0 = temp0 ^ state2[9];
		temp0 = temp0 ^ state2[12];
		temp0 = temp0 ^ state2[13];
		temp0 = temp0 ^ state2[14];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 4) ^ ROL32(temp0, 19) ^ injection_constants[i * 16 + 2];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[1];
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[3];
		temp1 = temp1 ^ state2[4];
		temp1 = temp1 ^ state2[6];
		temp1 = temp1 ^ state2[8];
		temp1 = temp1 ^ state2[10];
		temp1 = temp1 ^ state2[12];
		temp1 = temp1 ^ state2[13];
		temp1 = temp1 ^ state2[14];
		temp1 = temp1 ^ ROL32(temp1, 3) ^ ROL32(temp1, 14) ^ injection_constants[i * 16 + 3];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[3 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[3] = temp0 + temp1;


		temp0 = 0;
		temp0 = temp0 ^ state2[1];
		temp0 = temp0 ^ state2[2];
		temp0 = temp0 ^ state2[3];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[5];
		temp0 = temp0 ^ state2[7];
		temp0 = temp0 ^ state2[9];
		temp0 = temp0 ^ state2[11];
		temp0 = temp0 ^ state2[13];
		temp0 = temp0 ^ state2[14];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 27) ^ ROL32(temp0, 31) ^ injection_constants[i * 16 + 4];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[3];
		temp1 = temp1 ^ state2[7];
		temp1 = temp1 ^ state2[8];
		temp1 = temp1 ^ state2[10];
		temp1 = temp1 ^ state2[14];
		temp1 = temp1 ^ ROL32(temp1, 3) ^ ROL32(temp1, 8) ^ injection_constants[i * 16 + 5];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[5 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[5] = temp0 + temp1;


		temp0 = 0;
		temp0 = temp0 ^ state2[1];
		temp0 = temp0 ^ state2[3];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[8];
		temp0 = temp0 ^ state2[9];
		temp0 = temp0 ^ state2[11];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 17) ^ ROL32(temp0, 26) ^ injection_constants[i * 16 + 6];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[6];
		temp1 = temp1 ^ state2[7];
		temp1 = temp1 ^ state2[9];
		temp1 = temp1 ^ state2[10];
		temp1 = temp1 ^ state2[15];
		temp1 = temp1 ^ ROL32(temp1, 3) ^ ROL32(temp1, 12) ^ injection_constants[i * 16 + 7];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[7 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[7] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state2[0];
		temp0 = temp0 ^ state2[1];
		temp0 = temp0 ^ state2[3];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[5];
		temp0 = temp0 ^ state2[6];
		temp0 = temp0 ^ state2[8];
		temp0 = temp0 ^ state2[10];
		temp0 = temp0 ^ state2[11];
		temp0 = temp0 ^ state2[12];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 18) ^ ROL32(temp0, 22) ^ injection_constants[i * 16 + 8];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[1];
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[9];
		temp1 = temp1 ^ state2[11];
		temp1 = temp1 ^ state2[13];
		temp1 = temp1 ^ state2[15];
		temp1 = temp1 ^ ROL32(temp1, 12) ^ ROL32(temp1, 18) ^ injection_constants[i * 16 + 9];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[9 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[9] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state2[0];
		temp0 = temp0 ^ state2[1];
		temp0 = temp0 ^ state2[2];
		temp0 = temp0 ^ state2[3];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[5];
		temp0 = temp0 ^ state2[6];
		temp0 = temp0 ^ state2[7];
		temp0 = temp0 ^ state2[10];
		temp0 = temp0 ^ state2[14];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 4) ^ ROL32(temp0, 7) ^ injection_constants[i * 16 + 10];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[1];
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[3];
		temp1 = temp1 ^ state2[8];
		temp1 = temp1 ^ state2[11];
		temp1 = temp1 ^ state2[12];
		temp1 = temp1 ^ ROL32(temp1, 4) ^ ROL32(temp1, 31) ^ injection_constants[i * 16 + 11];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[11 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[11] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state2[1];
		temp0 = temp0 ^ state2[2];
		temp0 = temp0 ^ state2[3];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[9];
		temp0 = temp0 ^ state2[12];
		temp0 = temp0 ^ state2[13];
		temp0 = temp0 ^ ROL32(temp0, 12) ^ ROL32(temp0, 27) ^ injection_constants[i * 16 + 12];

		temp1 = 0;
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[3];
		temp1 = temp1 ^ state2[4];
		temp1 = temp1 ^ state2[5];
		temp1 = temp1 ^ state2[10];
		temp1 = temp1 ^ state2[13];
		temp1 = temp1 ^ state2[14];
		temp1 = temp1 ^ ROL32(temp1, 7) ^ ROL32(temp1, 17) ^ injection_constants[i * 16 + 13];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[13 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[13] = temp0 + temp1;

		temp0 = 0;
		temp0 = temp0 ^ state2[3];
		temp0 = temp0 ^ state2[4];
		temp0 = temp0 ^ state2[5];
		temp0 = temp0 ^ state2[6];
		temp0 = temp0 ^ state2[11];
		temp0 = temp0 ^ state2[14];
		temp0 = temp0 ^ state2[15];
		temp0 = temp0 ^ ROL32(temp0, 7) ^ ROL32(temp0, 8) ^ injection_constants[i * 16 + 14];

		temp1 = 0;
		temp1 = temp1 ^ state2[0];
		temp1 = temp1 ^ state2[1];
		temp1 = temp1 ^ state2[2];
		temp1 = temp1 ^ state2[3];
		temp1 = temp1 ^ state2[5];
		temp1 = temp1 ^ state2[7];
		temp1 = temp1 ^ state2[8];
		temp1 = temp1 ^ state2[9];
		temp1 = temp1 ^ state2[10];
		temp1 = temp1 ^ state2[11];
		temp1 = temp1 ^ state2[15];
		temp1 = temp1 ^ ROL32(temp1, 1) ^ ROL32(temp1, 13) ^ injection_constants[i * 16 + 15];
		temp0 = temp0 + temp1;
		temp0 = ROL32(temp0, 8);
		state[15 - 1] = temp0;
		temp1 = ROL32(temp1, 24);
		state[15] = temp0 + temp1;
	}
	if (Hash_Rounds & 1)
	{
		for (j = 0; j < 16; ++j) //可以和前面的循环整合
		{
			state[j] = state2[j];
		}
	}

}

void EaglesongSponge(unsigned char * output, unsigned int output_length, const unsigned char * input, unsigned int input_length, unsigned char delimiter,unsigned int salt)
{
	uint32_t state[16];
	uint32_t i, j, k;

	uint32_t integer;

	// initialize to zero
	for (i = 0; i < 16; ++i) //
	{
		state[i] = 0;
	}
	// absorbing
	for (i = 0; i < ((input_length + 1) * 8 + Hash_Rate - 1) / Hash_Rate; ++i) //循环结构疑似过于繁琐
	{
		for (j = 0; j < Hash_Rate / 32; ++j)
		{
			integer = 0;
			for (k = 0; k < 4; ++k)
			{
				if (i*Hash_Rate / 8 + j * 4 + k < input_length)
				{
					if (i == 0 && j == 0)
						integer = (integer << 8) ^ ((salt >> (8 * k)) & 0xff) ^ input[i*(Hash_Rate / 8)+j * 4 + k];
					else 
						integer = (integer << 8) ^ input[i*(Hash_Rate / 8)+j * 4 + k];
				}
				else if (i*Hash_Rate / 8 + j * 4 + k == input_length)
				{
					integer = (integer << 8) ^ delimiter;//
				}
			}
			state[j] = state[j] ^ integer;
		}
		EaglesongPermutation(state);
	}
	// squeezing
	for (i = 0; i < output_length / (Hash_Rate / 8); ++i)
	{
		for (j = 0; j < Hash_Rate / 32; ++j)
		{
			for (k = 0; k < 4; ++k)
			{
				output[i*Hash_Rate / 8 + j * 4 + k] = (state[j] >> (8 * k)) & 0xff;//赋值给char类型不需要于0xff
			}
		}
//		EaglesongPermutation(state);//显然没有必要，如果是最后一次
	}
}

inline void EaglesongHash(unsigned char * output, const unsigned char * input, unsigned int input_length, unsigned int salt)
{
	EaglesongSponge(output, 32, input, input_length, 0x06, salt);
}

struct GPU_DEVICE
{
	uint8_t chain_block[MaxMesageLen];
	uint32_t salt_out;
	uint32_t salt_in;
	uint8_t target[Hash_Len];
	uint8_t  *gchain_block = NULL;
	uint8_t  *gtarget = NULL;
	uint32_t  *gsalt_out = NULL;
};

GPU_DEVICE *gpu_divices[MaxGpuNum] = {0};
uint32_t gpu_divices_cnt = 0;

__global__ void eaglesong(uint8_t *chain_block, uint32_t blacklen, uint32_t salt, uint32_t *gsalt_out, uint8_t* target)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;

	uint32_t block_tid = id % threadsPerBlock;
	uint32_t temp, i, j, k, u, integer, blockid, flag, id_salt;
	uint32_t stat0, stat1, stat2, stat3, stat4, stat5, stat6, stat7, stat8, stat9, stat10, stat11, stat12, stat13, stat14, stat15;

	__shared__ uint8_t chain_block_Byte[Data_Len];//同线程访问同地址，只要一个周期,由于长度不足一个bank,所以不同地址也是一个周期
	__shared__ uint8_t target_Byte[Hash_Len];//同线程访问同地址，只要一个周期,由于长度不足一个bank,所以不同地址也是一个周期

	stat0 = stat1 = stat2 = stat3 = stat4 = stat5 = stat6 = stat7 = stat8 = stat9 = stat10 = stat11 = stat12 = stat13 = stat14 = stat15 = 0;
	if (block_tid < blacklen)chain_block_Byte[block_tid] = chain_block[block_tid]; 
	if (block_tid < Hash_Len)target_Byte[block_tid] = target[block_tid];
	__syncthreads();

	id_salt = salt + id;
	for (blockid = 0; blockid < (((blacklen + 1) <<3) + Hash_Rate - 1) / Hash_Rate; ++blockid) //循环结构疑似过于繁琐
	{
		for (j = 0; j < Hash_Rate / 32; ++j)
		{
			integer = 0;
			for (k = 0; k < 4; ++k)
			{
				if (blockid*(Hash_Rate >>3) + (j << 2) + k < blacklen)
				{
					if (blockid == 0 && j == 0)//salt??????????????????????????????????????????????????? 这里必须想办法插入盐 45585280  5121941
						integer = (integer << 8) ^ ((id_salt >> (k << 3)) & 0xff) ^ chain_block_Byte[blockid*(Hash_Rate >> 3) + (j << 2) + k];
					else
						integer = (integer << 8) ^ chain_block_Byte[blockid*(Hash_Rate >> 3) + (j << 2) + k];
				}
				else if (blockid*(Hash_Rate >> 3) + (j << 2) + k == blacklen)
				{
					integer = (integer << 8) ^ 0x06;
				}
			}		
			switch (j)
			{
				case 0:stat0 ^= integer; break;
				case 1:stat1 ^= integer; break;
				case 2:stat2 ^= integer; break;
				case 3:stat3 ^= integer; break;
				case 4:stat4 ^= integer; break;
				case 5:stat5 ^= integer; break;
				case 6:stat6 ^= integer; break;
				case 7:stat7 ^= integer; break;
				case 8:stat8 ^= integer; break;
				case 9:stat9 ^= integer; break;
				case 10:stat10 ^= integer; break;
				case 11:stat11 ^= integer; break;
				case 12:stat12 ^= integer; break;
				case 13:stat13 ^= integer; break;
				case 14:stat14 ^= integer; break;
				case 15:stat15 ^= integer; break;
			}
		}


		for (i = 0; i < (Hash_Rounds<<4); i+=16)
		{
			ep_sta(i);
		}
	}

	flag = 0;


	for (j = 0; j < Hash_Rate / 32; ++j)
	{
		switch (j)
		{
			case 0: integer = stat0; break;
			case 1: integer = stat1; break;
			case 2: integer = stat2; break;
			case 3: integer = stat3; break;
			case 4: integer = stat4; break;
			case 5: integer = stat5; break;
			case 6: integer = stat6; break;
			case 7: integer = stat7; break;
			case 8: integer = stat8; break;
			case 9: integer = stat9; break;
			case 10:integer = stat10; break;
			case 11:integer = stat11; break;
			case 12:integer = stat12; break;
			case 13:integer = stat13; break;
			case 14:integer = stat14; break;
			case 15:integer = stat15; break;
		}

		for (i = 0; i < 4; ++i)
		{
			if (target_Byte[(j << 2) + i]>((integer >> (i << 3)) & 0xff))
			{
				flag = 1;
				break;
			}
			else	if (target_Byte[(j << 2) + i] < ((integer >> (i << 3)) & 0xff))break;
		}
		if (i != 4)break;
	}

	if (flag)atomicExch(gsalt_out, id_salt);
}

int gpu_hash(uint32_t gpuid)
{
	clock_t t1, t2;
	if (HashNum > MaxHashNum) {
		printf("HashNum out of bound!!!\n");
		return 0;
	}

	if (Data_Len > MaxMesageLen) {
		printf("Data_Len out of bound!!!\n");
		return 0;
	}

	if (gpu_divices[gpuid]->gchain_block == NULL)
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->gchain_block, sizeof(gpu_divices[gpuid]->chain_block)) != hipSuccess) {
			printf("gchain_block hipMalloc error\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->gsalt_out == NULL) 
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->gsalt_out, sizeof(gpu_divices[gpuid]->salt_out)) != hipSuccess) {
			printf("ghash_out hipMalloc error\n");
			return 0;
		}
	}
	if (gpu_divices[gpuid]->gtarget == NULL)
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->gtarget, sizeof(gpu_divices[gpuid]->target)) != hipSuccess) {
			printf("gtarget hipMalloc error\n");
			return 0;
		}
	}
	t1 = clock();
	if (hipMemcpy(gpu_divices[gpuid]->gchain_block, gpu_divices[gpuid]->chain_block, sizeof(gpu_divices[gpuid]->chain_block), hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("gchain_block copy memory error\n");
		return 0;
	}
	if (hipMemcpy(gpu_divices[gpuid]->gtarget, gpu_divices[gpuid]->target, sizeof(gpu_divices[gpuid]->target), hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("gtarget copy memory error\n");
		return 0;
	}
	eaglesong << <HashNum / threadsPerBlock, threadsPerBlock >> >(gpu_divices[gpuid]->gchain_block, Data_Len, gpu_divices[gpuid]->salt_in, gpu_divices[gpuid]->gsalt_out, gpu_divices[gpuid]->gtarget);
	hipDeviceSynchronize();

	if (hipMemcpy(&(gpu_divices[gpuid]->salt_out), gpu_divices[gpuid]->gsalt_out, sizeof(gpu_divices[gpuid]->salt_out), hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("ghash_out copy memory error\n");
		return 0;
	}
	hipDeviceSynchronize();
	t2 = clock();
	printf("time: %f  %f ms %f per sec\n", (double)(t2 - t1) / CLOCKS_PER_SEC, (double)(t2 - t1) / (CLOCKS_PER_SEC / 1000.0) / HashNum, HashNum / ((double)(t2 - t1) / CLOCKS_PER_SEC));

	printf("salt = %5d\n", gpu_divices[gpuid]->salt_out);

	return gpu_divices[gpuid]->salt_out;
}

GPU_DEVICE* New_GPU_DEVICE()
{
	GPU_DEVICE* p = NULL;
	p = (GPU_DEVICE*)malloc(sizeof(GPU_DEVICE));
	if (p != NULL)
	{
		p->gsalt_out = NULL;
		p->gchain_block = NULL;
		p->gtarget = NULL;
	}
	return p;
}
void RESET_GPU_DEVICE(uint32_t gpuid)
{
	
	memset(gpu_divices[gpuid]->chain_block, 0, sizeof(gpu_divices[gpuid]->chain_block));
	gpu_divices[gpuid]->salt_out = 0;
	gpu_divices[gpuid]->salt_in = 0;

	hipFree(gpu_divices[gpuid]->gsalt_out);
	hipFree(gpu_divices[gpuid]->gchain_block);
	hipFree(gpu_divices[gpuid]->gtarget);

	gpu_divices[gpuid]->gsalt_out = NULL;
	gpu_divices[gpuid]->gchain_block = NULL;
	gpu_divices[gpuid]->gtarget = NULL;
}

void GPU_Count()
{
	hipError_t cudaStatus;
	int num;
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceCount(&num);
	printf("deviceCount := %d\n", num);
	gpu_divices_cnt = 0;
	for (int i = 0; i<num; i++)
	{

		hipGetDeviceProperties(&prop, i);
		printf("name:%s\n", prop.name);
		printf("totalGlobalMem:%d GB\n", prop.totalGlobalMem / 1024 / 1024 / 1024);
		printf("multiProcessorCount:%d\n", prop.multiProcessorCount);
		printf("maxThreadsPerBlock:%d\n", prop.maxThreadsPerBlock);
		printf("sharedMemPerBlock:%d KB\n", prop.sharedMemPerBlock/1024);
		printf("major:%d,minor:%d\n", prop.major, prop.minor);
		gpu_divices_cnt++;
	}
	if (gpu_divices_cnt > MaxGpuNum)gpu_divices_cnt = MaxGpuNum;
}
unsigned char input[MaxMesageLen] = "Hello, world!\nHello, world!\nHello, world!\nHello, world!\nHello, world!\nHello, world!\nHello, world!\n";
unsigned char output[Hash_Len];
int main(int argc, char ** argv)
{
	int c, i, j, sum = 100000, temp0;
	clock_t t1, t2;
	unsigned int salt;
	for (i = 0; i < sum; i++)EaglesongHash(output, input, Data_Len, i);
	GPU_Count();
	for (i = 0; i < gpu_divices_cnt;i++)
	{
		hipSetDevice(i);
		gpu_divices[i] = New_GPU_DEVICE();
		memcpy(gpu_divices[i]->chain_block, input, MaxMesageLen);
		memset(gpu_divices[i]->target, 0xff, sizeof(gpu_divices[i]->target));
		gpu_divices[i]->target[0] = 0;
		gpu_divices[i]->target[1] = 0;
		gpu_divices[i]->target[2] = 0;
		gpu_divices[i]->target[3] = 0x07;//这个不是百分之百找到，均值为1
		gpu_divices[i]->salt_in = 0;
		salt = gpu_hash(i);
	}

	EaglesongHash(output, input, Data_Len, salt);
	printf("salt = %5d\n", salt);
	for (i = 0; i < 32; ++i)
	{
		printf("%02x", output[i]);
	}
	return 0;
}